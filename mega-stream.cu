/*

  Copyright 2016 Tom Deakin, University of Bristol

  This file is part of mega-stream.

  mega-stream is free software: you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation, either version 3 of the License, or
  (at your option) any later version.

  mega-stream is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU General Public License for more details.

  You should have received a copy of the GNU General Public License
  along with mega-stream.  If not, see <http://www.gnu.org/licenses/>.


  This aims to test the theory that streaming many large arrays causes memory
  bandwidth limits not to be reached, and latency becomes a dominating factor.
  We run a kernel with a similar form to the original triad, but with more than
  3 input arrays.

  The main kernel computes:
  r(i,j,k) = q(i,j,k) + a(i)*x(i,j) + b(i)*y(i,j) + c(i)*z(i,j)
  sum(j,k) = SUM(r(:,j,k))
*/

#define VERSION "0.2.1"


#include <hip/hip_runtime.h>
#include <float.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>

#define MIN(a,b) ((a) < (b)) ? (a) : (b)
#define MAX(a,b) ((a) > (b)) ? (a) : (b)

#define IDX2(i,j,ni) ((i)+(ni)*(j))
#define IDX3(i,j,k,ni,nj) ((i)+(ni)*IDX2((j),(k),(nj)))

/*
  Arrays are defined in terms of 3 sizes
  The large arrays are of size SMALL*MEDIUM*LARGE and are indexed with 3 indicies.
  The medium arrays are of size SMALL*MEDIUM and are indexed with 2 indicies.
  The small arrays are of size SMALL and are indexed with 1 index.

  By default the large array has 2^27 elements, and the small array has 64 elements (2^6).
*/
#define LARGE  4096 // 2^12
#define MEDIUM  512 // 2^9
#define SMALL    64 // 2^6

/* Default alignment of 2 MB page boundaries */
#define ALIGNMENT 2*1024*1024

/* Tollerance with which to check final array values */
#define TOLR 1.0E-15

void parse_args(int argc, char *argv[]);

int L_size = LARGE;
int M_size = MEDIUM;
int S_size = SMALL;
int ntimes = 100;

void kernel(
  const int S_size, const int M_size, const int L_size,
  double *r, double *q, double *x, double *y, double *z,
  double *a, double *b, double *c, double *sum
)
{
  /**************************************************************************
   * Kernel
   *************************************************************************/
  for (int k = 0; k < L_size; k++)
  {
    for (int j = 0; j < M_size; j++)
    {
      double total = 0.0;
      for (int i = 0; i < S_size; i++)
      {
        r[IDX3(i,j,k,S_size,M_size)] =
          q[IDX3(i,j,k,S_size,M_size)]
          + a[i] * x[IDX2(i,j,S_size)]
          + b[i] * y[IDX2(i,j,S_size)]
          + c[i] * z[IDX2(i,j,S_size)];

        total += r[IDX3(i,j,k,S_size,M_size)];
      }
      sum[IDX2(j,k,M_size)] += total;
    }
  }
}
 
int main(int argc, char *argv[])
{

  printf("MEGA-STREAM! - v%s\n\n", VERSION);


  parse_args(argc, argv);

  printf("Small arrays:  %d elements\t\t(%.1lf KB)\n",
    S_size, S_size*sizeof(double)*1.0E-3);

  printf("Medium arrays: %d x %d elements\t(%.1lf MB)\n",
    S_size, M_size, S_size*M_size*sizeof(double)*1.0E-6);

  printf("Large arrays:  %d x %d x %d elements\t(%.1lf MB)\n",
    S_size, M_size, L_size, S_size*M_size*L_size*sizeof(double)*1.0E-6);

  const double footprint = (double)sizeof(double) * 1.0E-6 * (
    2.0*L_size*M_size*S_size +   /* r, q */
    3.0*M_size*S_size +          /* x, y, z */
    3.0*S_size +                 /* a, b, c */
    L_size*M_size                /* sum */
    );
  printf("Memory footprint: %.1lf MB\n", footprint);

  /* Total memory moved - the arrays plus an extra sum as update is += */
  const double size = footprint + (double)sizeof(double) * L_size*M_size * 1.0E-6;

  printf("Running %d times\n", ntimes);

  printf("\n");

  double timings[ntimes];


  double *q = (double*)malloc(sizeof(double)*L_size*M_size*S_size);
  double *r = (double*)malloc(sizeof(double)*L_size*M_size*S_size);

  double *x = (double*)malloc(sizeof(double)*M_size*S_size);
  double *y = (double*)malloc(sizeof(double)*M_size*S_size);
  double *z = (double*)malloc(sizeof(double)*M_size*S_size);

  double *a = (double*)malloc(sizeof(double)*S_size);
  double *b = (double*)malloc(sizeof(double)*S_size);
  double *c = (double*)malloc(sizeof(double)*S_size);

  double *sum = (double*)malloc(sizeof(double)*L_size*M_size);

  /* Initalise the data */
  {
    for (int k = 0; k < L_size; k++)
    {
      for (int j = 0; j < M_size; j++)
      {
        for (int i = 0; i < S_size; i++)
        {
          q[IDX3(i,j,k,S_size,M_size)] = 0.1;
          r[IDX3(i,j,k,S_size,M_size)] = 0.0;
        }
      }
    }

    for (int j = 0; j < M_size; j++)
    {
      for (int i = 0; i < S_size; i++)
      {
        x[IDX2(i,j,S_size)] = 0.2;
        y[IDX2(i,j,S_size)] = 0.3;
        z[IDX2(i,j,S_size)] = 0.4;
      }
    }

    for (int i = 0; i < S_size; i++)
    {
      a[i] = 0.6;
      b[i] = 0.7;
      c[i] = 0.8;
    }

    for (int k = 0; k < L_size; k++)
    {
      for (int j = 0; j < M_size; j++)
      {
        sum[IDX2(j,k,M_size)] = 0.0;
      }
    }
  }

  /* Run the kernel multiple times */
  for (int t = 0; t < ntimes; t++)
  {
    struct timeval tick;
    gettimeofday(&tick, 0);

    kernel(S_size, M_size, L_size, r, q, x, y, z, a, b, c, sum);

    struct timeval tock;
    gettimeofday(&tock, 0);
    timings[t] = (1.0E6*(tock.tv_sec-tick.tv_sec) + tock.tv_usec-tick.tv_usec)/1.0E3;

  }

  /* Check the results */
  const double gold = 0.1 + 0.2*0.6 + 0.3*0.7 + 0.4*0.8;
  const double gold_sum = gold*S_size*ntimes;

  /* Check the r array */
  for (int k = 0; k < L_size; k++)
    for (int j = 0; j < M_size; j++)
      for (int i = 0; i < S_size; i++)
      {
        if (fabs(r[IDX3(i,j,k,S_size,M_size)]-gold) > TOLR)
        {
          printf("Results incorrect - at (%d,%d,%d), %lf should be %lf\n",
            i,j,k, r[IDX3(i,j,k,S_size,M_size)], gold);
          goto sumcheck;
        }
      }

sumcheck:
  /* Check the reduction array */
  for (int i = 0; i < L_size*M_size; i++)
  {
    if (fabs(sum[i]-gold_sum) > TOLR)
    {
      printf("Reduction incorrect - at %d, %lf should be %lf\n",
        i, sum[i], gold_sum);
      break;
    }
  }

  /* Print timings */
  double min = DBL_MAX;
  double max = 0.0;
  double avg = 0.0;
  for (int t = 1; t < ntimes; t++)
  {
    min = MIN(min, timings[t]);
    max = MAX(max, timings[t]);
    avg += timings[t];
  }
  avg /= (double)(ntimes - 1);

  printf("Bandwidth MB/s  Min time    Max time    Avg time\n");
  printf("%12.1f %11.6f %11.6f %11.6f\n", size/min, min, max, avg);

  /* Free memory */
  free(q);
  free(r);
  free(x);
  free(y);
  free(z);
  free(a);
  free(b);
  free(c);
  free(sum);

  return EXIT_SUCCESS;

}

void parse_args(int argc, char *argv[])
{
  for (int i = 1; i < argc; i++)
  {
    if (strcmp(argv[i], "--large") == 0)
    {
      L_size = atoi(argv[++i]);
    }
    else if (strcmp(argv[i], "--medium") == 0)
    {
      M_size = atoi(argv[++i]);
    }
    else if (strcmp(argv[i], "--small") == 0)
    {
      S_size = atoi(argv[++i]);
    }
    else if (strcmp(argv[i], "--swap") == 0)
    {
      int tmp = L_size;
      L_size = M_size;
      M_size = tmp;
    }
    else if (strcmp(argv[i], "--ntimes") == 0)
    {
      ntimes = atoi(argv[++i]);
      if (ntimes < 2)
      {
        fprintf(stderr, "ntimes must be 2 or greater\n");
        exit(EXIT_FAILURE);
      }
    }
    else if (strcmp(argv[i], "--help") == 0)
    {
      printf("Usage: %s [OPTION]\n", argv[0]);
      printf("\t --large n \tSet size of large dimension\n");
      printf("\t --medium n \tSet size of medium dimension\n");
      printf("\t --small n \tSet size of small dimension\n");
      printf("\t --swap\tSwap medium and large sizes over\n");
      printf("\t --ntimes n\tRun the benchmark n times\n");
      printf("\n");
      printf("\t Large  is %12d elements\n", LARGE);
      printf("\t Medium is %12d elements\n", MEDIUM);
      printf("\t Small  is %12d elements\n", SMALL);
      exit(EXIT_SUCCESS);
    }
    else
    {
      fprintf(stderr, "Unrecognised argument \"%s\"\n", argv[i]);
      exit(EXIT_FAILURE);
    }
  }
}

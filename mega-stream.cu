/*

  Copyright 2016 Tom Deakin, University of Bristol

  This file is part of mega-stream.

  mega-stream is free software: you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation, either version 3 of the License, or
  (at your option) any later version.

  mega-stream is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU General Public License for more details.

  You should have received a copy of the GNU General Public License
  along with mega-stream.  If not, see <http://www.gnu.org/licenses/>.


  This aims to test the theory that streaming many large arrays causes memory
  bandwidth limits not to be reached, and latency becomes a dominating factor.
  We run a kernel with a similar form to the original triad, but with more than
  3 input arrays.

  The main kernel computes:
  r(i,j,k) = q(i,j,k) + a(i)*x(i,j) + b(i)*y(i,j) + c(i)*z(i,j)
  sum(j,k) = SUM(r(:,j,k))
*/

#define VERSION "0.2.1"


#include <hip/hip_runtime.h>
#include <float.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>

#define MIN(a,b) ((a) < (b)) ? (a) : (b)
#define MAX(a,b) ((a) > (b)) ? (a) : (b)

#define IDX2(i,j,ni) ((i)+(ni)*(j))
#define IDX3(i,j,k,ni,nj) ((i)+(ni)*IDX2((j),(k),(nj)))

/*
  Arrays are defined in terms of 3 sizes
  The large arrays are of size SMALL*MEDIUM*LARGE and are indexed with 3 indicies.
  The medium arrays are of size SMALL*MEDIUM and are indexed with 2 indicies.
  The small arrays are of size SMALL and are indexed with 1 index.

  By default the large array has 2^27 elements, and the small array has 64 elements (2^6).
*/
#define LARGE  4096 // 2^12
#define MEDIUM  512 // 2^9
#define SMALL    64 // 2^6

/* Default alignment of 2 MB page boundaries */
#define ALIGNMENT 2*1024*1024

/* Tollerance with which to check final array values */
#define TOLR 1.0E-15

void parse_args(int argc, char *argv[]);
void check_error(int line);

int L_size = LARGE;
int M_size = MEDIUM;
int S_size = SMALL;
int ntimes = 100;
bool split = false;

__global__
void kernel(
  const int S_size, const int M_size, const int L_size,
  double * __restrict__ r,
  const double * __restrict__ q,
  const double * __restrict__ x,
  const double * __restrict__ y,
  const double * __restrict__ z,
  const double * __restrict__ a,
  const double * __restrict__ b,
  const double * __restrict__ c,
  double * __restrict__ sum
)
{
  /**************************************************************************
   * Kernel
   *************************************************************************/
  const int k = blockIdx.x / M_size;
  const int j = blockIdx.x % M_size;
  const int i = threadIdx.x;

  extern __shared__ double totals[];

  r[IDX3(i,j,k,S_size,M_size)] =
    q[IDX3(i,j,k,S_size,M_size)]
    + a[i] * x[IDX2(i,j,S_size)]
    + b[i] * y[IDX2(i,j,S_size)]
    + c[i] * z[IDX2(i,j,S_size)];

  totals[i] = r[IDX3(i,j,k,S_size,M_size)];

  for (int offset = blockDim.x / 2; offset > 0; offset /= 2)
  {
    __syncthreads();
    if (i < offset)
    {
      totals[i] += + totals[i + offset];
    }
  }

  if (i == 0)
    sum[IDX2(j,k,M_size)] += totals[i];
}
 
__global__
void kernel_compute(
  const int S_size, const int M_size, const int L_size,
  double * __restrict__ r,
  const double * __restrict__ q,
  const double * __restrict__ x,
  const double * __restrict__ y,
  const double * __restrict__ z,
  const double * __restrict__ a,
  const double * __restrict__ b,
  const double * __restrict__ c
)
{
  const int id = threadIdx.x + blockIdx.x * blockDim.x;
  const int i = id % S_size;
  const int j = (id / S_size) % M_size;
  const int k = (id / S_size) / M_size;

  r[IDX3(i,j,k,S_size,M_size)] =
    q[IDX3(i,j,k,S_size,M_size)]
    + a[i] * x[IDX2(i,j,S_size)]
    + b[i] * y[IDX2(i,j,S_size)]
    + c[i] * z[IDX2(i,j,S_size)];
}

__global__
void kernel_reduce(
  const int S_size, const int M_size, const int L_size,
  const double * __restrict__ r,
  double * __restrict__ sum
)
{
  const int k = blockIdx.x / M_size;
  const int j = blockIdx.x % M_size;
  const int i = threadIdx.x;

  extern __shared__ double totals[];

  totals[i] = r[IDX3(i,j,k,S_size,M_size)];

  for (int offset = blockDim.x / 2; offset > 0; offset /= 2)
  {
    __syncthreads();
    if (i < offset)
    {
      totals[i] += + totals[i + offset];
    }
  }

  if (i == 0)
    sum[IDX2(j,k,M_size)] += totals[i];

}

int main(int argc, char *argv[])
{

  printf("MEGA-STREAM! - v%s\n\n", VERSION);


  parse_args(argc, argv);

  printf("Small arrays:  %d elements\t\t(%.1lf KB)\n",
    S_size, S_size*sizeof(double)*1.0E-3);

  printf("Medium arrays: %d x %d elements\t(%.1lf MB)\n",
    S_size, M_size, S_size*M_size*sizeof(double)*1.0E-6);

  printf("Large arrays:  %d x %d x %d elements\t(%.1lf MB)\n",
    S_size, M_size, L_size, S_size*M_size*L_size*sizeof(double)*1.0E-6);

  const double footprint = (double)sizeof(double) * 1.0E-6 * (
    2.0*L_size*M_size*S_size +   /* r, q */
    3.0*M_size*S_size +          /* x, y, z */
    3.0*S_size +                 /* a, b, c */
    L_size*M_size                /* sum */
    );
  printf("Memory footprint: %.1lf MB\n", footprint);

  /* Total memory moved - the arrays plus an extra sum as update is += */
  const double size = footprint + (double)sizeof(double) * L_size*M_size * 1.0E-6;

  printf("Running %d times\n", ntimes);

  printf("\n");

  double timings[ntimes];


  double *q = (double*)malloc(sizeof(double)*L_size*M_size*S_size);
  double *r = (double*)malloc(sizeof(double)*L_size*M_size*S_size);

  double *x = (double*)malloc(sizeof(double)*M_size*S_size);
  double *y = (double*)malloc(sizeof(double)*M_size*S_size);
  double *z = (double*)malloc(sizeof(double)*M_size*S_size);

  double *a = (double*)malloc(sizeof(double)*S_size);
  double *b = (double*)malloc(sizeof(double)*S_size);
  double *c = (double*)malloc(sizeof(double)*S_size);

  double *sum = (double*)malloc(sizeof(double)*L_size*M_size);

  /* Initalise the data */
  {
    for (int k = 0; k < L_size; k++)
    {
      for (int j = 0; j < M_size; j++)
      {
        for (int i = 0; i < S_size; i++)
        {
          q[IDX3(i,j,k,S_size,M_size)] = 0.1;
          r[IDX3(i,j,k,S_size,M_size)] = 0.0;
        }
      }
    }

    for (int j = 0; j < M_size; j++)
    {
      for (int i = 0; i < S_size; i++)
      {
        x[IDX2(i,j,S_size)] = 0.2;
        y[IDX2(i,j,S_size)] = 0.3;
        z[IDX2(i,j,S_size)] = 0.4;
      }
    }

    for (int i = 0; i < S_size; i++)
    {
      a[i] = 0.6;
      b[i] = 0.7;
      c[i] = 0.8;
    }

    for (int k = 0; k < L_size; k++)
    {
      for (int j = 0; j < M_size; j++)
      {
        sum[IDX2(j,k,M_size)] = 0.0;
      }
    }
  }

  /* Device memory */
  double *d_r, *d_q, *d_x, *d_y, *d_z, *d_a, *d_b, *d_c, *d_sum;
  hipMalloc(&d_r, sizeof(double)*L_size*M_size*S_size);
  check_error(__LINE__);
  hipMalloc(&d_q, sizeof(double)*L_size*M_size*S_size);
  check_error(__LINE__);
  hipMalloc(&d_x, sizeof(double)*M_size*S_size);
  check_error(__LINE__);
  hipMalloc(&d_y, sizeof(double)*M_size*S_size);
  check_error(__LINE__);
  hipMalloc(&d_z, sizeof(double)*M_size*S_size);
  check_error(__LINE__);
  hipMalloc(&d_a, sizeof(double)*S_size);
  check_error(__LINE__);
  hipMalloc(&d_b, sizeof(double)*S_size);
  check_error(__LINE__);
  hipMalloc(&d_c, sizeof(double)*S_size);
  check_error(__LINE__);
  hipMalloc(&d_sum, sizeof(double)*M_size*L_size);
  check_error(__LINE__);

  hipMemcpy(d_r, r, sizeof(double)*L_size*M_size*S_size, hipMemcpyHostToDevice);
  check_error(__LINE__);
  hipMemcpy(d_q, q, sizeof(double)*L_size*M_size*S_size, hipMemcpyHostToDevice);
  check_error(__LINE__);
  hipMemcpy(d_x, x, sizeof(double)*M_size*S_size, hipMemcpyHostToDevice);
  check_error(__LINE__);
  hipMemcpy(d_y, y, sizeof(double)*M_size*S_size, hipMemcpyHostToDevice);
  check_error(__LINE__);
  hipMemcpy(d_z, z, sizeof(double)*M_size*S_size, hipMemcpyHostToDevice);
  check_error(__LINE__);
  hipMemcpy(d_a, a, sizeof(double)*S_size, hipMemcpyHostToDevice);
  check_error(__LINE__);
  hipMemcpy(d_b, b, sizeof(double)*S_size, hipMemcpyHostToDevice);
  check_error(__LINE__);
  hipMemcpy(d_c, c, sizeof(double)*S_size, hipMemcpyHostToDevice);
  check_error(__LINE__);
  hipMemcpy(d_sum, sum, sizeof(double)*M_size*L_size, hipMemcpyHostToDevice);
  check_error(__LINE__);

  /* Run the kernel multiple times */
  for (int t = 0; t < ntimes; t++)
  {
    struct timeval timstr;
    gettimeofday(&timstr, 0);
    double tick = timstr.tv_sec + (timstr.tv_usec / 1000000.0);

    int blocks = M_size*L_size;
    int threads = S_size;

    if (!split)
    {
      kernel<<<blocks, threads, sizeof(double)*S_size>>>(S_size, M_size, L_size, d_r, d_q, d_x, d_y, d_z, d_a, d_b, d_c, d_sum);
      check_error(__LINE__);
    }
    else
    {
      int workers = 256;
      int work = (L_size*M_size*S_size) / workers;

      kernel_compute<<<work, workers>>>(S_size, M_size, L_size, d_r, d_q, d_x, d_y, d_z, d_a, d_b, d_c);
      check_error(__LINE__);

      kernel_reduce<<<blocks, threads, sizeof(double)*S_size>>>(S_size, M_size, L_size, d_r, d_sum);
      check_error(__LINE__);
    }

    hipDeviceSynchronize();
    check_error(__LINE__);

    gettimeofday(&timstr, 0);
    double tock = timstr.tv_sec + (timstr.tv_usec / 1000000.0);

    timings[t] = tock-tick;

  }

  /* Check the results */
  const double gold = 0.1 + 0.2*0.6 + 0.3*0.7 + 0.4*0.8;
  const double gold_sum = gold*S_size*ntimes;

  /* Copy back memory */
  hipMemcpy(r, d_r, sizeof(double)*L_size*M_size*S_size, hipMemcpyDeviceToHost);
  check_error(__LINE__);
  hipMemcpy(q, d_q, sizeof(double)*L_size*M_size*S_size, hipMemcpyDeviceToHost);
  check_error(__LINE__);
  hipMemcpy(x, d_x, sizeof(double)*M_size*S_size, hipMemcpyDeviceToHost);
  check_error(__LINE__);
  hipMemcpy(y, d_y, sizeof(double)*M_size*S_size, hipMemcpyDeviceToHost);
  check_error(__LINE__);
  hipMemcpy(z, d_z, sizeof(double)*M_size*S_size, hipMemcpyDeviceToHost);
  check_error(__LINE__);
  hipMemcpy(a, d_a, sizeof(double)*S_size, hipMemcpyDeviceToHost);
  check_error(__LINE__);
  hipMemcpy(b, d_b, sizeof(double)*S_size, hipMemcpyDeviceToHost);
  check_error(__LINE__);
  hipMemcpy(c, d_c, sizeof(double)*S_size, hipMemcpyDeviceToHost);
  check_error(__LINE__);
  hipMemcpy(sum, d_sum, sizeof(double)*M_size*L_size, hipMemcpyDeviceToHost);
  check_error(__LINE__);

  /* Check the r array */
  for (int k = 0; k < L_size; k++)
    for (int j = 0; j < M_size; j++)
      for (int i = 0; i < S_size; i++)
      {
        if (fabs(r[IDX3(i,j,k,S_size,M_size)]-gold) > TOLR)
        {
          printf("Results incorrect - at (%d,%d,%d), %lf should be %lf\n",
            i,j,k, r[IDX3(i,j,k,S_size,M_size)], gold);
          goto sumcheck;
        }
      }

sumcheck:
  /* Check the reduction array */
  for (int i = 0; i < L_size*M_size; i++)
  {
    if (fabs(sum[i]-gold_sum) > TOLR)
    {
      printf("Reduction incorrect - at %d, %lf should be %lf\n",
        i, sum[i], gold_sum);
      break;
    }
  }

  /* Print timings */
  double min = DBL_MAX;
  double max = 0.0;
  double avg = 0.0;
  for (int t = 1; t < ntimes; t++)
  {
    min = MIN(min, timings[t]);
    max = MAX(max, timings[t]);
    avg += timings[t];
  }
  avg /= (double)(ntimes - 1);

  printf("Bandwidth MB/s  Min time    Max time    Avg time\n");
  printf("%12.1f %11.6f %11.6f %11.6f\n", size/min, min, max, avg);

  /* Free memory */
  hipFree(d_q);
  hipFree(d_r);
  hipFree(d_x);
  hipFree(d_y);
  hipFree(d_z);
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
  hipFree(d_sum);
  free(q);
  free(r);
  free(x);
  free(y);
  free(z);
  free(a);
  free(b);
  free(c);
  free(sum);

  return EXIT_SUCCESS;

}

void parse_args(int argc, char *argv[])
{
  for (int i = 1; i < argc; i++)
  {
    if (strcmp(argv[i], "--large") == 0)
    {
      L_size = atoi(argv[++i]);
    }
    else if (strcmp(argv[i], "--medium") == 0)
    {
      M_size = atoi(argv[++i]);
    }
    else if (strcmp(argv[i], "--small") == 0)
    {
      S_size = atoi(argv[++i]);
    }
    else if (strcmp(argv[i], "--swap") == 0)
    {
      int tmp = L_size;
      L_size = M_size;
      M_size = tmp;
    }
    else if (strcmp(argv[i], "--ntimes") == 0)
    {
      ntimes = atoi(argv[++i]);
      if (ntimes < 2)
      {
        fprintf(stderr, "ntimes must be 2 or greater\n");
        exit(EXIT_FAILURE);
      }
    }
    else if (strcmp(argv[i], "--split") == 0)
    {
      split = true;
      printf("Using split kernels\n");
    }
    else if (strcmp(argv[i], "--help") == 0)
    {
      printf("Usage: %s [OPTION]\n", argv[0]);
      printf("\t --large n \tSet size of large dimension\n");
      printf("\t --medium n \tSet size of medium dimension\n");
      printf("\t --small n \tSet size of small dimension\n");
      printf("\t --swap\tSwap medium and large sizes over\n");
      printf("\t --ntimes n\tRun the benchmark n times\n");
      printf("\t --split n\tSplit compute and reduction kernels\n");
      printf("\n");
      printf("\t Large  is %12d elements\n", LARGE);
      printf("\t Medium is %12d elements\n", MEDIUM);
      printf("\t Small  is %12d elements\n", SMALL);
      exit(EXIT_SUCCESS);
    }
    else
    {
      fprintf(stderr, "Unrecognised argument \"%s\"\n", argv[i]);
      exit(EXIT_FAILURE);
    }
  }
}

void check_error(int line)
{
  hipError_t err = hipGetLastError();
  if (err != hipSuccess)
  {
    fprintf(stderr, "Error on line %d: %s\n", line, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}

